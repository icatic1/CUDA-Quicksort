#include "hip/hip_runtime.h"
/*
 * CUDA-Quicksort.cu
 *
 * Copyright © 2012-2015 Emanuele Manca
 *
 **********************************************************************************************
 **********************************************************************************************
 *
	This file is part of CUDA-Quicksort.

	CUDA-Quicksort is free software: you can redistribute it and/or modify
	it under the terms of the GNU General Public License as published by
	the Free Software Foundation, either version 3 of the License, or
	(at your option) any later version.

	CUDA-Quicksort is distributed in the hope that it will be useful,
	but WITHOUT ANY WARRANTY; without even the implied warranty of
	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
	GNU General Public License for more details.

	You should have received a copy of the GNU General Public License
	along with CUDA-Quicksort.

	If not, see http://www.gnu.org/licenses/gpl-3.0.txt and http://www.gnu.org/copyleft/gpl.html


  **********************************************************************************************
  **********************************************************************************************
 *
 * Contact: Ing. Emanuele Manca
 *
 * Department of Electrical and Electronic Engineering,
 * University of Cagliari,
 * P.zza D’Armi, 09123, Cagliari, Italy
 *
 * email: emanuele.manca@diee.unica.it
 *
 *
 * This software contains source code provided by NVIDIA Corporation
 * license: http://developer.download.nvidia.com/licenses/general_license.txt
 *
 * this software uses the library of NVIDIA CUDA SDK and the Cederman and Tsigas' GPU Quick Sort
 *
 */



#include <thrust/scan.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include "scan.h"
#include <scan_common.h>
#include "CUDA-Quicksort.h"

 // extern __shared__ char sMemory[];


__device__ inline  double atomicMax(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int assumed;
	unsigned long long int old = *address_as_ull;

	assumed = old;
	old = atomicCAS(address_as_ull,
		assumed,
		__double_as_longlong(max(val, __longlong_as_double(assumed))));

	while (assumed != old)
	{
		assumed = old;
		old = atomicCAS(address_as_ull,
			assumed,
			__double_as_longlong(max(val, __longlong_as_double(assumed))));
	}
	return __longlong_as_double(old);
}


__device__ inline double atomicMin(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	assumed = old;
	old = atomicCAS(address_as_ull,
		assumed,
		__double_as_longlong(min(val, __longlong_as_double(assumed))));
	while (assumed != old)
	{
		assumed = old;
		old = atomicCAS(address_as_ull,
			assumed,
			__double_as_longlong(min(val, __longlong_as_double(assumed))));
	}
	return __longlong_as_double(old);
}





template <typename Type>
__device__ inline void Comparator(

	Type& valA,
	Type& valB,
	uint dir
) {
	Type t;
	if ((valA > valB) == dir) {
		t = valA; valA = valB; valB = t;
	}
}




static __device__ __forceinline__ unsigned int __qsflo(unsigned int word)
{
	unsigned int ret;
	asm volatile("bfind.u32 %0, %1;" : "=r"(ret) : "r"(word));
	return ret;
}

template <typename Type>
__global__ void globalBitonicSort(Type* indata, Type* outdata, Block<Type>* bucket, bool inputSelect, int shmem, int blocks)
{
	extern __shared__ uint shared[];


	Type* data;
	// 0 1 - 0 
	// 2 3 - 1 
	// 2 3 - 2 
	Block<Type> cord = bucket[blockIdx.x / blocks];
	uint tid = threadIdx.x + ((blockIdx.x % blocks) * blockDim.x);

	uint size = cord.end - cord.begin;
	bool select = !(cord.select);

	if (cord.end - cord.begin > shmem || cord.end - cord.begin == 0)
		return;

	unsigned int bitonicSize = 1 << (__qsflo(size - 1U) + 1);


	if (select)
		data = indata;
	else
		data = outdata;

	//__syncthreads();

	for (int i = tid;i < size;i += blockDim.x)
		shared[i] = data[i + cord.begin];


	for (int i = tid + size;i < bitonicSize;i += blockDim.x)
		shared[i] = 0xffffffff;

	__syncthreads();


	for (uint size = 2; size < bitonicSize; size <<= 1) {
		//Bitonic merge
		uint ddd = 1 ^ ((tid & (size / 2)) != 0);
		for (uint stride = size / 2; stride > 0; stride >>= 1) {
			__syncthreads();
			uint pos = 2 * tid - (tid & (stride - 1));
			//if(pos <bitonicSize){
			Comparator(
				shared[pos + 0],
				shared[pos + stride],
				ddd
			);
			// }
		}
	}


	//ddd == dir for the last bitonic merge step

	for (uint stride = bitonicSize / 2; stride > 0; stride >>= 1) {
		__syncthreads();
		uint pos = 2 * tid - (tid & (stride - 1));
		// if(pos <bitonicSize){
		Comparator(
			shared[pos + 0],
			shared[pos + stride],
			1
		);
		// }
	}

	__syncthreads();

	// Write back the sorted data to its correct position
	for (int i = tid;i < size;i += blockDim.x)
		indata[i + cord.begin] = shared[i];

}




template <typename Type>
__global__ void quick(Type* indata, Type* buffer, Partition<Type>* partition, Block<Type>* bucket, int shmem, int size)
{
	extern __shared__ char s[];
	Type* sh_out = (Type*)&s[128];

	// __shared__ uint start1,end1;
	// __shared__ uint left,right;

	uint* start1, * end1;
	uint* left, * right;

	start1 = (uint*)&s; end1 = (uint*)&s[sizeof(uint)];
	left = (uint*)&s[2 * sizeof(uint)]; right = (uint*)&s[3 * sizeof(uint)];

	// printf("Here also");
	// *left = 32;
	// printf("Here also also");

	int tix = threadIdx.x;

	uint start = partition[blockIdx.x].from;
	uint end = partition[blockIdx.x].end;
	Type pivot = partition[blockIdx.x].pivot;
	uint nseq = partition[blockIdx.x].ibucket;

	uint lo = 0;
	uint hi = 0;

	Type lmin = 0xffffffff;
	Type rmax = 0;

	Type d;
	Type dd;


	// start read on 1° tile and store the coordinates of the items that must
	// be moved on the left or on the right of the pivot

	/*lo = (((d < pivot) * (lo + 1) + (d >= pivot) * lo) * (ii < end)) + (lo * (ii >= end));
	hi = (((d <= pivot) * (hi)+(d > pivot) * (hi + 1)) * (ii < end)) + (hi * (ii >= end));*/
	/*lo = (((dd < pivot) * (lo + 1) + (dd >= pivot) * lo) * (ii < end)) + (lo * (ii >= end));
	hi = (((dd <= pivot) * (hi)+(dd > pivot) * (hi + 1)) * (ii < end)) + (hi * (ii >= end));*/

	uint ii = tix + start;
	bool endUslov = (ii < end);
	d = indata[ii * endUslov];
	lo += (d < pivot) && endUslov;
	hi += (d > pivot) && endUslov;
	lmin = d * endUslov + lmin * !endUslov;
	rmax = d * endUslov + rmax * !endUslov;

	ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) && endUslov;
	hi += (dd > pivot) && endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;

	ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) && endUslov;
	hi += (dd > pivot) && endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;

	ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) && endUslov;
	hi += (dd > pivot) && endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;

	// 4 + 4
	ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) && endUslov;
	hi += (dd > pivot) && endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;

	ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) && endUslov;
	hi += (dd > pivot) && endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;

	ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) && endUslov;
	hi += (dd > pivot) && endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;

	ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) && endUslov;
	hi += (dd > pivot) && endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;


	// 8 + 8
	/*ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) * endUslov;
	hi += (dd > pivot) * endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;

	ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) * endUslov;
	hi += (dd > pivot) * endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;

	ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) * endUslov;
	hi += (dd > pivot) * endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;

	ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) * endUslov;
	hi += (dd > pivot) * endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;

	ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) * endUslov;
	hi += (dd > pivot) * endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;

	ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) * endUslov;
	hi += (dd > pivot) * endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;

	ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) * endUslov;
	hi += (dd > pivot) * endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;

	ii += blockDim.x;
	endUslov = (ii < end);
	dd = indata[ii * endUslov];
	lo += (dd < pivot) * endUslov;
	hi += (dd > pivot) * endUslov;
	lmin = min(lmin, dd) * endUslov + lmin * !endUslov;
	rmax = max(rmax, dd) * endUslov + rmax * !endUslov;*/


	/*i++;
	d = indata[(tix + start + i * blockDim.x) * ((tix + start + i * blockDim.x) < end)];
	lo = ((d < pivot) * (lo + 1) + (d >= pivot) * lo) * ((tix + start + i * blockDim.x) < end) + lo * ((tix + start + i * blockDim.x) >= end);
	hi = ((d <= pivot) * (hi)+(d > pivot) * (hi + 1)) * ((tix + start + i * blockDim.x) < end) + lo * ((tix + start + i * blockDim.x) >= end);
	lmin = min(lmin, d);
	rmax = max(rmax, d);*/

	/*if (tix + start<end)
	{
		d = indata[tix+start];

		//count items smaller or bigger than the pivot
		// if d<pivot then ll++ else ll
		lo += (d < pivot);
		hi += (d > pivot);
		// lo=(d<pivot)*(lo+1)+(d>=pivot)*lo;
		// if d>pivot then lr++ else lr
		// hi=(d<=pivot)*(hi)+(d>pivot)*(hi+1);
		// lo = ((d < pivot) ? (lo + 1) : (lo));
		// hi = ((d > pivot) ? (hi + 1) : (hi));

		lmin = d;
		rmax= d;
	}*/

	/*i += blockDim.x;
	Type dd = indata[ii * (ii < end)];
	lo = (((dd < pivot) * (lo + 1) + (dd >= pivot) * lo) * (ii < end)) + (lo * (ii >= end));
	hi = (((dd <= pivot) * (hi)+(dd > pivot) * (hi + 1)) * (ii < end)) + (hi * (ii >= end));
	lmin = min(lmin, dd) * (ii < end) + lmin * (ii >= end);
	rmax = max(rmax, dd) * (ii < end) + rmax * (ii >= end);*/

	//read and store the coordinates on next tiles for each block
	/*for (uint i = tix + start + blockDim.x;i<end;i += blockDim.x)
	{
		Type d= indata[i];

		//count items smaller or bigger than the pivot
		// lo = ( d <  pivot ) *(lo+1) + ( d >= pivot )*lo;
		// hi = ( d <= pivot ) *(hi)   +  (d >  pivot )*(hi+1);
		lo += (d < pivot);
		hi += (d > pivot);
		// lo = ((d < pivot) ? (lo + 1) : (lo));
		// hi = ((d > pivot) ? (hi + 1) : (hi));

		//compute max and min of tile items
		lmin = min(lmin,d);
		rmax = max(rmax,d);

	}*/

	//compute max and min of every partition

	compareInclusive(rmax, lmin, (Type*)sh_out, blockDim.x);
	// printf("Hrer");
	__syncthreads();

	if (tix == blockDim.x - 1)
	{
		//compute absolute max and min for the bucket
		atomicMax(&bucket[nseq].maxPiv, rmax);
		atomicMin(&bucket[nseq].minPiv, lmin);
	}
	__syncthreads();


	/*
	 * calculate the coordinates of its assigned item to each thread,
	 * which are necessary to known in which subsequences the item must be copied
	 *
	 */

	scan1Inclusive2(lo, hi, (uint*)sh_out, blockDim.x);
	lo = lo - 1;
	hi = shmem - hi;

	if (tix == blockDim.x - 1)
	{
		*left = lo + 1;
		*right = shmem - hi;

		*start1 = atomicAdd(&bucket[nseq].nextbegin, *left);
		*end1 = atomicSub(&bucket[nseq].nextend, *right);
	}

	__syncthreads();

	//if (threadIdx.x == 0 && blockIdx.x == 0) {
	//	printf("%d, %d, %d < %d\n", lo, hi, d, pivot);
	//}

	/*if (threadIdx.x == 0 && blockIdx.x == 0) {
		// printf("%d, %d, %d, %d, %d\n", d, pivot, (iii < end), (d < pivot), ((iii < end) && (d < pivot)));
		printf("%d\n", sh_out[hi]);
		printf("%d\n", sh_out[lo]);
		printf("%d, %d, %d < %d\n", lo, hi, d, pivot);
	}*/

	//thread blocks write on the shared memory the items smaller and bigger than the first tile's pivot
	//uint iii = tix + start;
	//sh_out[lo] = d * (iii < end) * (d < pivot)/* + (sh_out[lo] * (iii >= end)) + (sh_out[lo] * ((iii < end) & (d >= pivot)))*/;
	//lo -= ((iii < end) & (d < pivot));
	//sh_out[hi] = d * (iii < end) * (d > pivot)/* + (sh_out[hi] * (iii >= end)) + (sh_out[hi] * ((iii < end) & (d <= pivot)))*/;
	//hi += ((iii < end) & (d > pivot));*/

	// for (int i = 0; i < shmem; i += blockDim.x) {
	//	sh_out[i] = 0;
	//}

	/*uint iii = tix + start;
	sh_out[lo] = d * (iii < end) * (d < pivot) + (sh_out[lo] * (iii >= end)) + (sh_out[lo] * ((iii < end) & (d >= pivot)));
	lo -= (iii < end) * (d < pivot);
	sh_out[hi] = d * (iii < end) * (d > pivot) + (sh_out[hi] * (iii >= end)) + (sh_out[hi] * ((iii < end) & (d <= pivot)));
	hi += (iii < end) * (d > pivot);*/
	// uint iii = tix + start;
	// uint loIndex = lo * (iii < end) * (d < pivot) + 

	/*uint iii = tix + start;
	bool lo1 = ((iii < end) && (d < pivot));
	// uint loIndex = lo * lo1 + (shmem + tix) * !lo1;
	bool hi1 = ((iii < end) && (d > pivot));
	// uint hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	uint index = lo * lo1 + hi * hi1 + ((shmem + tix) * !lo1 * !hi1);
	// sh_out[loIndex] = d * lo1;
	sh_out[index] = d;
	lo -= lo1;
	// sh_out[hiIndex] = d * hi1;
	hi += hi1;*/

	uint iii = tix + start;
	bool lo1 = ((iii < end) && (d < pivot));
	bool hi1 = ((iii < end) && (d > pivot));
	uint index = lo * lo1 + hi * hi1 + ((shmem + tix) * !lo1 * !hi1);
	sh_out[index] = d;
	lo -= lo1;
	hi += hi1;

	/*if (tix + start<end)
	{
		//items smaller than pivot
		if(d<pivot)
			{sh_out[lo]=d; lo--;}

		//items bigger than pivot
		if(d>pivot)
			{sh_out[hi]=d; hi++;}
	}*/

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	hi1 = ((iii < end) && (dd > pivot));
	index = lo * lo1 + hi * hi1 + ((shmem + tix) * !lo1 * !hi1);
	sh_out[index] = dd;
	lo -= lo1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	hi1 = ((iii < end) && (dd > pivot));
	index = lo * lo1 + hi * hi1 + ((shmem + tix) * !lo1 * !hi1);
	sh_out[index] = dd;
	lo -= lo1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	hi1 = ((iii < end) && (dd > pivot));
	index = lo * lo1 + hi * hi1 + ((shmem + tix) * !lo1 * !hi1);
	sh_out[index] = dd;
	lo -= lo1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	hi1 = ((iii < end) && (dd > pivot));
	index = lo * lo1 + hi * hi1 + ((shmem + tix) * !lo1 * !hi1);
	sh_out[index] = dd;
	lo -= lo1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	hi1 = ((iii < end) && (dd > pivot));
	index = lo * lo1 + hi * hi1 + ((shmem + tix) * !lo1 * !hi1);
	sh_out[index] = dd;
	lo -= lo1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	hi1 = ((iii < end) && (dd > pivot));
	index = lo * lo1 + hi * hi1 + ((shmem + tix) * !lo1 * !hi1);
	sh_out[index] = dd;
	lo -= lo1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	hi1 = ((iii < end) && (dd > pivot));
	index = lo * lo1 + hi * hi1 + ((shmem + tix) * !lo1 * !hi1);
	sh_out[index] = dd;
	lo -= lo1;
	hi += hi1;

	/*iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;

	// 4 + 4
	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;*/


	// 8 + 8
	/*iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;

	iii += blockDim.x;
	dd = indata[iii];
	lo1 = ((iii < end) && (dd < pivot));
	loIndex = lo * lo1 + (shmem + tix) * !lo1;
	hi1 = ((iii < end) && (dd > pivot));
	hiIndex = hi * hi1 + (shmem + tix) * !hi1;
	sh_out[loIndex] = dd * lo1;
	lo -= lo1;
	sh_out[hiIndex] = dd * hi1;
	hi += hi1;*/

	//thread blocks write on the shared memory the items smaller and bigger than next tiles' pivot
	/*for (uint i = start + tix + blockDim.x;i<end;i += blockDim.x)
	{

		Type d=indata[i];
		//items smaller than the pivot
		if(d<pivot)
			{sh_out[lo--]=d;}

		//items bigger than the pivot
		if(d>pivot)
			{sh_out[hi++]=d;}


	}*/

	__syncthreads();
	iii = tix;
	bool buffer1 = (iii < (*left)); bool buffer2 = iii >= shmem - (*right);
	int bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	// 4 + 4
	iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	// 8 + 8
	/*iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];

	iii += blockDim.x;
	buffer1 = (iii < (*left)); buffer2 = iii >= shmem - (*right);
	bufferIndex = ((*start1) + iii) * buffer1 + ((*end1) + iii - shmem) * buffer2 + ((size + threadIdx.x) * !buffer1 * !buffer2);
	buffer[bufferIndex] = sh_out[iii];*/



	//items smaller and bigger than the pivot already sorted in the shared memory are coalesced written on the global memory
	//partial results of each thread block stored on the shared memory are merged together in two subsequences within the global memory
	//coalesced writing of next tiles on the global memory
	/*for (uint i = tix;i<shmem;i += blockDim.x)
	{
		if (i<*left)
			buffer[*start1+i]=sh_out[i];

		if(i>=shmem-*right)
			buffer[*end1+i-shmem]=sh_out[i];
	}*/

}



//this function assigns the attributes to each partition of each bucket
//a thread block is assigned to a specific partition
template <typename Type>
__global__ void partitionAssign(struct Block<Type>* bucket, uint* npartitions, struct Partition<Type>* partition, int shmem, int partitionSize, int nbucket)
{
	int tx = threadIdx.x;
	int bx = blockIdx.x;

	uint beg = bucket[bx].nextbegin;
	uint end = bucket[bx].nextend;
	Type pivot = bucket[bx].pivot;
	uint from;
	uint to;

	bool uslov = bx > 0;
	from = npartitions[(bx - 1) * uslov] * uslov;
	to = npartitions[bx];
	/*if (bx>0)
	{
		from=npartitions[bx-1];
		to=npartitions[bx];
	}
	else
	{
		from=0;
		to=npartitions[bx];
	}*/


	uint i = tx + from;


	/*uslov = i < to;
	uint begin = (beg + shmem * tx);
	int partitionIndex = (i * uslov) + ((partitionSize) * !uslov);
	partition[partitionIndex].from = begin * uslov;
	partition[partitionIndex].end = (begin + shmem) * uslov;
	partition[partitionIndex].pivot= pivot * uslov;
	partition[partitionIndex].ibucket= bx * uslov + (nbucket + 1) * !uslov;*/

	if (i < to)
	{
		uint begin = beg + shmem * tx;
		partition[i].from = begin;
		partition[i].end = begin + shmem;
		partition[i].pivot = pivot;
		partition[i].ibucket = bx;

	}


	for (uint i = tx + from + blockDim.x;i < to;i += blockDim.x)
	{
		uint begin = beg + shmem * (i - from);
		partition[i].from = begin;
		partition[i].end = begin + shmem;
		partition[i].pivot = pivot;
		partition[i].ibucket = bx;
	}
	__syncthreads();
	if (tx == 0 && to - from > 0) partition[to - 1].end = end;


}

//this function enters the pivot value in the central bucket's items
template <typename Type>
__global__ void insertPivot(Type* data, struct Block<Type>* bucket, int nbucket)
{

	Type pivot = bucket[blockIdx.x].pivot;
	uint start = bucket[blockIdx.x].nextbegin;
	uint end = bucket[blockIdx.x].nextend;
	bool is_altered = bucket[blockIdx.x].done;

	if (is_altered && blockIdx.x < nbucket)
		for (uint j = start + threadIdx.x; j < end; j += blockDim.x)
			data[j] = pivot;


}


//this function assigns the new attributes of each bucket
template <typename Type>
__global__ void bucketAssign(Block<Type>* bucket, uint* npartitions, int nbucket, int select, int shmem, int minSize)
{

	uint i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < nbucket) {
		bool is_altered = bucket[i].done;
		if (is_altered)
		{
			//read on i node
			uint orgbeg = bucket[i].begin;
			uint from = bucket[i].nextbegin;
			uint orgend = bucket[i].end;
			uint end = bucket[i].nextend;
			Type pivot = bucket[i].pivot;
			Type minPiv = bucket[i].minPiv;
			Type maxPiv = bucket[i].maxPiv;

			//compare each bucket's max and min to the pivot
			Type lmaxpiv = min(pivot, maxPiv);
			Type rminpiv = max(pivot, minPiv);

			//write on i+nbucket node
			bucket[i + nbucket].begin = orgbeg;
			bucket[i + nbucket].nextbegin = orgbeg;
			bucket[i + nbucket].nextend = from;
			bucket[i + nbucket].end = from;
			bucket[i + nbucket].pivot = (minPiv + lmaxpiv) / 2;

			//if(select)
			//	bucket[i+nbucket].done   = (from-orgbeg)>1024;// && (minPiv!=maxPiv);
			//else
			bucket[i + nbucket].done = (from - orgbeg) > (minSize / 2) && (minPiv != maxPiv);
			bucket[i + nbucket].select = select;
			bucket[i + nbucket].minPiv = 0xffffffffffffffff;
			bucket[i + nbucket].maxPiv = 0;
			//bucket[i+nbucket].finish=false;

			//calculate the number of partitions (npartitions) necessary to the i+nbucket bucket
			/*if (!bucket[i + nbucket].done)
				 npartitions[i+nbucket] = 0;
			else npartitions[i+nbucket] = (from-orgbeg+shmem-1)/shmem;*/
			npartitions[i + nbucket] = bucket[i + nbucket].done * (from - orgbeg + shmem - 1) / shmem;

			//write on i node
			bucket[i].begin = end;
			bucket[i].nextbegin = end;
			bucket[i].nextend = orgend;
			bucket[i].pivot = (rminpiv + maxPiv) / 2 + 1;

			//if(select)
				//bucket[i].done   = (orgend-end)>1024;// && (minPiv!=maxPiv);
			//	else
			bucket[i].done = (orgend - end) > (minSize / 2) && (minPiv != maxPiv);
			bucket[i].select = select;
			bucket[i].minPiv = 0xffffffffffffffff;
			bucket[i].maxPiv = 0;
			//bucket[i].finish=false;

			//calculate the number of partitions (npartitions) necessary to the i-bucket bucket
			npartitions[i] = bucket[i].done * (orgend - end + shmem - 1) / shmem;
			/*if (!bucket[i].done)
				npartitions[i]=0;
			else
				npartitions[i]=(orgend-end+shmem-1)/shmem;*/

		}
	}


}



template <typename Type>
__global__ void init(Type* data, Block<Type>* bucket, uint* npartitions, int size, int nblocks)
{
	uint i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < nblocks)
	{
		bucket[i].nextbegin = 0;
		bucket[i].begin = 0;

		bucket[i].nextend = 0 + size * (i == 0);
		bucket[i].end = 0 + size * (i == 0);
		npartitions[i] = 0;
		bucket[i].done = false + i == 0;
		bucket[i].select = false;
		bucket[i].maxPiv = 0x0;
		bucket[i].minPiv = 0xffffffffffffffff;
		bucket[i].pivot = 0 + (i == 0) * ((min(min(data[0], data[size / 2]), data[size - 1]) + max(max(data[0], data[size / 2]), data[size - 1])) / 2);
	}

}



template <typename Type>
void sort(Type* inputData, Type* outputData, uint size, uint threadCount, int device, double* wallClock)
{
	int shmem = threadCount * 8;
	printf("%d\n", shmem);
	hipSetDevice(device);

	hipGetLastError();
	//hipDeviceReset();

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);

	StopWatchInterface* htimer = NULL;
	StopWatchInterface* btimer = NULL;
	Type* ddata;
	Type* dbuffer;

	Block<Type>* dbucket;
	struct Partition<Type>* partition;
	uint* npartitions1, * npartitions2;

	uint* cudaBlocks = (uint*)malloc(4);

	uint blocks = (size + shmem - 1) / shmem;
	int partition_max = 262144;
	uint partitions = 2 * blocks * sizeof(Partition<Type>);

	unsigned long long int total = partition_max * sizeof(Block<Type>) + blocks * sizeof(Partition<Type>) + 2 * partition_max * sizeof(uint) + 3 * (size) * sizeof(Type);

	// printf("%d\n", deviceProp.sharedMemPerBlock);
	printf("\nINFO: Device Memory consumed is %.3f GB out of %.3f GB of available memory\n", ((double)total / GIGA), (double)deviceProp.totalGlobalMem / GIGA);

	//Allocating and initializing CUDA arrays
	sdkCreateTimer(&htimer);
	sdkCreateTimer(&btimer);
	checkCudaErrors(hipMalloc((void**)&dbucket, partition_max * sizeof(Block<Type>)));
	checkCudaErrors(hipMalloc((void**)&partition, partitions + 4 * sizeof(Partition<Type>))); //nblock


	checkCudaErrors(hipMalloc((void**)&npartitions1, partition_max * sizeof(uint)));
	checkCudaErrors(hipMalloc((void**)&npartitions2, partition_max * sizeof(uint)));

	checkCudaErrors(hipMalloc((void**)&dbuffer, (size) * sizeof(Type) + threadCount * sizeof(Type)));
	checkCudaErrors(hipMalloc((void**)&ddata, (size) * sizeof(Type) + threadCount * sizeof(Type)));

	checkCudaErrors(hipMemcpy(ddata, inputData, size * sizeof(Type), hipMemcpyHostToDevice));

	initScan();

	//setting GPU Cache
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(init<Type>), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(insertPivot<Type>), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(bucketAssign<Type>), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(partitionAssign<Type>), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(quick<Type>), hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(globalBitonicSort<Type>), hipFuncCachePreferShared);


	// 128 threads - 2048 elements, bucket limit - 4096, threads for bitonic 2048 
	uint maxBucket = 4 * shmem > 2048 ? 2048 : 4 * shmem;
	uint maxThreadBucket = maxBucket / 2 > 1024 ? 1024 : maxBucket / 2;
	uint bitonicBlockMult = (maxBucket / 2048) > 0 ? (maxBucket / 2048) : 1;

	checkCudaErrors(hipDeviceSynchronize());
	sdkResetTimer(&htimer);
	sdkResetTimer(&btimer);
	sdkStartTimer(&htimer);

	//initializing bucket array: initial attributes for each bucket
	init<Type> << <(2 * blocks + 255) / 256, 256 >> > (ddata, dbucket, npartitions1, size, partition_max);

	// uint maxBucket = ((2 * shmem > 2048) ? (2048) : (shmem));
	uint nbucket = 1;
	uint numIterations = 0;
	bool inputSelect = true;

	*cudaBlocks = blocks;
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("init() execution FAILED\n");
	checkCudaErrors(hipMemcpy(&npartitions2[0], cudaBlocks, sizeof(uint), hipMemcpyHostToDevice));


	// beginning of the first phase
	// this phase goes on until the size of the buckets is comparable to the SHARED_LIMIT size
	while (1)
	{

		/*
		 *       	---------------------    Pre-processing: Partitioning    ---------------------
		 *
		 * buckets are further divided in partitions based on their size
		 * the number of partitions needed for each subsequence is determined by the number of elements which can be
		 * processed by each thread block.
		 *
		 * the number of partitions (npartitions) for each block will depend on the shared memory size (SHARED_LIMIT)
		 *
		 */

		if (numIterations > 0)
		{	//1024 is the shared memory limit of scanInclusiveShort()
			if (nbucket <= 1024)
				scanInclusiveShort(npartitions2, npartitions1, 1, nbucket);
			else
				scanInclusiveLarge(npartitions2, npartitions1, 1, nbucket);

			checkCudaErrors(hipMemcpy(cudaBlocks, &npartitions2[nbucket - 1], sizeof(uint), hipMemcpyDeviceToHost));
		}

		if (*cudaBlocks == 0)
			break;


		/*
		 *  ---------------------     step 1    ---------------------
		 *
		 * 	A thread block is assigned to each different partition
		 * 	each partition is assigned coordinates, pivot and ....
		 */


		partitionAssign<Type> << <nbucket, 1024 >> > (dbucket, npartitions2, partition, shmem, partitions, nbucket);
		hipDeviceSynchronize();
		getLastCudaError("partitionAssign() execution FAILED\n");

		/*
			 ---------------------    step 2a    ---------------------

			 in this function each thread block creates two subsequences
			 to divide the items in the partition whose value is lower than
			 the pivot value, from the items whose value is higher than the pivot value
		 */

		if (inputSelect)
			quick<Type> << <*cudaBlocks, threadCount, 2 * sizeof(Type) * shmem + 256 >> > (ddata, dbuffer, partition, dbucket, shmem, size);
		else
			quick<Type> << <*cudaBlocks, threadCount, 2 * sizeof(Type) * shmem + 256 >> > (dbuffer, ddata, partition, dbucket, shmem, size);
		hipDeviceSynchronize();
		getLastCudaError("quick() execution FAILED\n");

		//step 2b: this function enters the pivot value in the central bucket's items
		insertPivot<Type> << <nbucket, 512 >> > (ddata, dbucket, nbucket);


		//step 3: parameters are assigned, linked to the two new buckets created in step 2
		bucketAssign<Type> << <(nbucket + 255) / 256, 256 >> > (dbucket, npartitions1, nbucket, inputSelect, shmem, maxBucket);
		hipDeviceSynchronize();
		getLastCudaError("insertPivot() or bucketAssign() execution FAILED\n");

		nbucket *= 2;

		inputSelect = !inputSelect;
		numIterations++;
		printf("%d--", numIterations);

		if (nbucket > (deviceProp.maxGridSize[0]))
			break;
		// if(numIterations==9) break;
	}

	/*
	 * start second phase:
	 * now the size of the buckets is such that they can be entirely processed by a thread block
	 *
	 */

	sdkStopTimer(&htimer);
	*wallClock = sdkGetTimerValue(&htimer);
	printf("\nIteracija: %d\n", numIterations);
	printf("Quicksort exec time: %f\n", sdkGetTimerValue(&htimer));

	sdkStartTimer(&btimer);
	if (nbucket > deviceProp.maxGridSize[0])
		fprintf(stderr, "ERROR: CUDA-Quicksort can't terminate sorting as the block threads needed to finish it are more than the Maximum x-dimension of FERMI GPU thread blocks. Please use Kepler GPUs as the Maximum x-dimension of their thread blocks is much higher\n");
	else
		globalBitonicSort<Type> << <bitonicBlockMult * nbucket, maxThreadBucket, sizeof(uint)* maxBucket >> > (ddata, dbuffer, dbucket, inputSelect, maxBucket, bitonicBlockMult);

	hipDeviceSynchronize();
	sdkStopTimer(&btimer);
	printf("Bitonic sort exec time: %f\n\n", sdkGetTimerValue(&btimer));

	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("globalBitonicSort() execution FAILED\n");




	// Copy the final result to the CPU in the outputData array
	checkCudaErrors(hipMemcpy(outputData, ddata, size * sizeof(Type), hipMemcpyDeviceToHost));

	// release resources
	checkCudaErrors(hipFree(ddata));
	checkCudaErrors(hipFree(dbuffer));
	checkCudaErrors(hipFree(dbucket));
	checkCudaErrors(hipFree(npartitions2));
	checkCudaErrors(hipFree(npartitions1));
	free(cudaBlocks);

	closeScan();
	return;
}



extern "C"
void CUDA_Quicksort(uint * inputData, uint * outputData, uint dataSize, uint threadCount, int Device, double* wallClock)
{

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, Device);

	if (deviceProp.major < 2)
	{
		fprintf(stderr, "Error: the GPU device %d has a Compute Capability of %d.%d, while a Compute Capability of 2.x is required to run the code\n",
			Device, deviceProp.major, deviceProp.minor);

		int deviceCount;
		hipGetDeviceCount(&deviceCount);

		fprintf(stderr, "       the Host system has the following GPU devices:\n");

		for (int device = 0; device < deviceCount; device++) {

			fprintf(stderr, "\t  the GPU device %d is a %s, with Compute Capability %d.%d\n",
				device, deviceProp.name, deviceProp.major, deviceProp.minor);
		}

		return;
	}

	sort<uint>(inputData, outputData, dataSize, threadCount, Device, wallClock);
}

extern "C"
void CUDA_Quicksort_64(double* inputData, double* outputData, uint dataSize, uint threadCount, int Device, double* wallClock)
{

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, Device);

	if (deviceProp.major < 2)
	{
		fprintf(stderr, "Error: the GPU device %d has a Compute Capability of %d.%d, while a Compute Capability of 2.x is required to run the code\n",
			Device, deviceProp.major, deviceProp.minor);

		int deviceCount;
		hipGetDeviceCount(&deviceCount);

		fprintf(stderr, "       the Host system has the following GPU devices:\n");

		for (int device = 0; device < deviceCount; device++) {

			fprintf(stderr, "\t  the GPU device %d is a %s, with Compute Capability %d.%d\n",
				device, deviceProp.name, deviceProp.major, deviceProp.minor);
		}

		return;
	}

	sort<double>(inputData, outputData, dataSize, threadCount, Device, wallClock);

}
